#include "hip/hip_runtime.h"

#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include ""
#include <opencv2/core/core.hpp>
#include <stdlib.h>
#include <stddef.h>
#include<ctime>

using namespace cv;
using namespace std;
#include <string>

#define WINDOW_SIZE (5)
#define FILTER_SIZE (WINDOW_SIZE*WINDOW_SIZE)
#define TILE_SIZE 16
#define MEDIAN_INDEX (FILTER_SIZE/2 +1)


__global__ void medianFilterKernel(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth, int imageHeight)
{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char filterVector[FILTER_SIZE];   //Take fiter window
	if ((row == 0) || (col == 0) || (row == imageHeight - 1) || (col == imageWidth - 1))
		outputImagekernel[row*imageWidth + col] = 0; //Deal with boundry conditions
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) {
			for (int y = 0; y < WINDOW_SIZE; y++) {
				filterVector[x*WINDOW_SIZE + y] = inputImageKernel[(row + x - 1)*imageWidth + (col + y - 1)];   // setup the filterign window.
			}
		}
		for (int i = 0; i < FILTER_SIZE; i++) {
			for (int j = i + 1; j < FILTER_SIZE; j++) {
				if (filterVector[i] > filterVector[j]) {
					//Swap the variables.
					char tmp = filterVector[i];
					filterVector[i] = filterVector[j];
					filterVector[j] = tmp;
				}
			}
		}
		outputImagekernel[row*imageWidth + col] = filterVector[12];   //Set the output variables.
	}
}

unsigned char* createImageBuffer(unsigned int bytes, unsigned char **devicePtr)
{
	unsigned char *ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
	hipHostGetDevicePointer(devicePtr, ptr, 0);
	return ptr;
}


int main(int argc, char** argv)
{
	/*VideoCapture camera(0);
	Mat frame;
	if (!camera.isOpened())
		return -1;

	camera >> frame;*/
	//IplImage* img = cvLoadImage("lena.jpg", 1);
	//IplImage* dst = cvCreateImage(cvGetSize(img), IPL_DEPTH_8U, 3);
	//double a[9] = { 1.0, 2.0, 1.0,
	//	0.0, 0.0, 0.0,
	//	-1.0, -2.0, -1.0};
	//CvMat k;
	//cvInitMatHeader(&k, 3, 3, CV_64FC1, a);

	//cvFilter2D(img, dst, &k, cvPoint(-1, -1));
	//cvSaveImage("filtered.jpg", dst);

	//namedWindow("Before", CV_WINDOW_AUTOSIZE);

	Mat src = imread("lenaSzum.jpeg", 1);

	Mat dst;


	//imshow("Before", src);

	//medianBlur(src, dst, 7);

	//imshow("Median filter", dst);


	int width = src.size().width;
	int height = src.size().height;

	unsigned char *sourceDataDevice, *filteredDataDevice;
	Mat source(src.size(), CV_8U, createImageBuffer(width * height, &sourceDataDevice));
	Mat filtered(src.size(), CV_8U, createImageBuffer(width * height, &filteredDataDevice));

	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)width / (float)TILE_SIZE),
		(int)ceil((float)height / (float)TILE_SIZE));

		cvtColor(src, source, CV_BGR2GRAY);
		clock_t start = clock();
		medianBlur(source, dst, 3);
		clock_t end = clock();
		double time = double(end - start) / ((double)CLOCKS_PER_SEC / 1000);
		cout << "Czas filtracji na CPU: " << time << "ms\n";

		start = clock();

		medianFilterKernel << <dimGrid, dimBlock >> > (sourceDataDevice, filteredDataDevice, width, height);
		//hipDeviceSynchronize();
		hipDeviceSynchronize();

		end = clock();
		time = double(end - start) / ((double)CLOCKS_PER_SEC / 1000);
		cout << "Czas filtracji na GPU: " << time << "ms\n";

		imshow("source", source);
		imshow("filtered", filtered);
		imshow("filtered with opencv", dst);
		waitKey();
}
